
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <cmath>

#define L 32	// lattice length; must be even number!
#define D 3 	// dimensions
#define N 32768 // number of spins on square lattice (N=L^D)
#define Q 2 	// number of Potts states

#define N_SWEEPS	100000

/* Distribute N/2 spins over threads and blocks:
   since we do "black" and "white" spins seperately, we only need half the threads:
   N_BLOCKS * N_THREADS_PER_BLOCK = N/2 */
#define N_BLOCKS			128   // number of blocks
#define N_THREADS_PER_BLOCK	128  // number of threads per block


/**********************************
 *  GPU RANDOM NUMBER GENERATION  *
 **********************************/

__device__ unsigned Tausworthe88(unsigned &z1, unsigned &z2, unsigned &z3)
{
	// Three-step generator with period 2^88
	unsigned b = (((z1 << 13) ^ z1) >> 19);
	z1 = (((z1 & 4294967294) << 12) ^ b);

	b = (((z2 << 2) ^ z2) >> 25);
	z2 = (((z2 & 4294967288) << 4)  ^ b);

	b = (((z3 << 3) ^ z3) >> 11);
	z3 = (((z3 & 4294967280) << 17) ^ b);

	return z1 ^ z2 ^ z3;
}

__device__ unsigned LCRNG(unsigned &z)  
{  
	const unsigned a = 1664525, c = 1013904223;
	return z = a * z + c;
}

__device__ float TauswortheLCRNG(unsigned &z1, unsigned &z2, unsigned &z3, unsigned &z)
{
	// combine both generators and normalize 0...2^32 to 0...1
	return (Tausworthe88(z1, z2, z3) ^ LCRNG(z)) * 2.3283064365e-10;
}


/******************************
 *  GPU METROPOLIS ALGORITHM  *
 ******************************/

#ifndef CONTRT
	__global__ void runMetropolis(int *seeds, unsigned short* spins, unsigned* neighbors, unsigned* spinIdList, 
						int* energyDifferences, int* magnetization, float beta)
#else
	__global__ void runMetropolis(int *seeds, unsigned short* spins, unsigned* neighbors, unsigned* spinIdList, 
						int* energyDifferences, int* magnetization, bool* contrt, float beta)
#endif
{
	const unsigned id = blockDim.x*blockIdx.x + threadIdx.x;

	// spin id from list of black or white spins:
	const unsigned spinId = spinIdList[id];

	// get seed values:
	unsigned z1 = seeds[4*id    ];	// Tausworthe seeds
	unsigned z2 = seeds[4*id + 1];
	unsigned z3 = seeds[4*id + 2];
	unsigned z  = seeds[4*id + 3];	// LCRNG seed

	// energy differences for this block:
	__shared__ int deltaE[N_THREADS_PER_BLOCK];
	__shared__ int blockSpins[N_THREADS_PER_BLOCK];
	deltaE[threadIdx.x] = 0;

	unsigned short spinstate = spins[spinId];	// get spin state from DRAM
	unsigned short nb[2*D];						// neighbor states

	#ifdef CONTRT
        bool nl[2 * D];
    #endif
	
	// get neighbor states:
	for(unsigned n=0; n<2*D; n++){
		nb[n] = spins[neighbors[2*D*spinId + n]];

		#ifdef CONTRT
            nl[n] = contrt[neighbors[2 * D * spinId + n]];
        #endif
	}

	// propose random new spin state:
	unsigned short newstate = floor(TauswortheLCRNG(z1, z2, z3, z) * Q);

	// energy difference: E'-E
	int E_before = 0;
	int E_after  = 0;

	for(unsigned short n=0; n<2*D; n++)
	{
		#ifdef CONTRT
			#ifndef STRONG
           		if((contrt[spinId] || nl[n]) && n > 3)
					continue;
			#else
				if((contrt[spinId] || nl[n]) && n > 3){
					if(spinstate == nb[n])
						E_before += 0.5;
		
					if(newstate == nb[n])
						E_after += 0.5;
					continue;
				}
			#endif
        #endif

		if(spinstate == nb[n])
			E_before++;
		
		if(newstate == nb[n])
			E_after++;
	}

	// acceptance probability:
	float dE = __int2float_rn(E_before - E_after);
	float pAccept = __expf(-beta*dE);

	if(TauswortheLCRNG(z1, z2, z3, z) <= pAccept)
	{
		spins[spinId] = newstate;   // flip spin
		spinstate = newstate;

		deltaE[threadIdx.x] = E_before - E_after;	// note energy difference
	}

	// remember locally in block for calculation of magnetization:
	blockSpins[threadIdx.x] = spinstate;

	// store new seed values in DRAM:
	seeds[4*id    ] = z1;	// Tausworthe seeds
	seeds[4*id + 1] = z2;
	seeds[4*id + 2] = z3;
	seeds[4*id + 3] = z;	// LCRNG seed


	__syncthreads();


	// sum up this block's energy delta and magnetization:
	if(threadIdx.x == 0)
	{
		int blockEnergyDiff = 0;
		int m = 0;

		for(unsigned i=0; i<blockDim.x; i++)
		{
			blockEnergyDiff += deltaE[i];

			if(blockSpins[i] == 0)
				m -= 1;
			else
				m += 1;
		}

		energyDifferences[blockIdx.x] += blockEnergyDiff;
		magnetization[blockIdx.x] = m;
	}
}



/******************************
 *  HOST FUNCTION (CPU PART)  *
 ******************************/

int main(int argc, char const **argv)
{
	int args = 1;

    #ifdef CONTRT
        int x = (argc > args)?(atoi(argv[args])):(5);
        args++; 

        int num_atom = (argc > args)?(atoi(argv[args])):(3);
        args++;
    #endif
    
    srand48(time(NULL));

	// each spin has value 0,..,Q-1
	unsigned short spins[N];

	#ifdef CONTRT
        bool contrt[N];
    #endif

	// calculate lattice volume elements:
	unsigned volume[D];
	for(unsigned i=0; i<=D; i++)
	{
		if(i == 0)
			volume[i] = 1;
		else
			volume[i] = volume[i-1] * L;
	}	


	/* Determine the "checkerboard color" (black or white) for each site and
	   initialise lattice with random spin states: */
	unsigned w=0, b=0;
	unsigned white[N/2], black[N/2];	// store ids of white/black sites

	for(unsigned i=0; i<N; i++)
	{
		// Sum of all coordinates even or odd? -> gives checkerboard color
		int csum = 0;
		for(int k=D-1; k>=0; k--)
			csum += ceil((i+1.0)/volume[k]) - 1;

		if((csum%2) == 0)	// white
		{
			white[w] = i;
			w++;
		}
		else				// black
		{
			black[b] = i;
			b++;
		}

		// random spin state:
		#ifdef RANDSTATE
			spins[i] = floor(drand48() * Q);
		#else
			spins[i] = 1;
		#endif

		#ifdef CONTRT
            contrt[i] = false;
        #endif
	}


	#ifdef CONTRT
        int n_ele = int(N / x);
    	for(unsigned elem = 0; elem < n_ele; elem++){
        	int n = 0;
        	while(n < num_atom){
            	unsigned index = static_cast<unsigned>(floor(x * drand48()) + elem * x);
            	if(!contrt[index]){
            		//std::cout << index << '\t' << contrt[index] << std::endl;
           			contrt[index] = true;
           			n++;
				}            	
        	}
    	}

        bool* devPtrContrt;
        cudaMalloc((void**)&devPtrContrt, sizeof(contrt));
        cudaMemcpy(devPtrContrt, &contrt, sizeof(contrt), cudaMemcpyHostToDevice);
    #endif

	// neighborhood table:
	unsigned neighbors[2*D*N];

	// calculate neighborhood table:
	for(unsigned i=0; i<N; i++)
	{
		unsigned short c=0;

		for(unsigned short dim=0; dim<D; dim++)	// dimension loop
		{
			for(short dir=-1; dir<=1; dir+=2)	// two directions in each dimension
			{
				// neighbor's id in spin list:
				int npos = i + dir * volume[dim];

				// periodic boundary conditions:
				int test = (i % volume[dim+1]) + dir*volume[dim];

				if(test < 0)
					npos += volume[dim+1];
				else if(test >= volume[dim+1])
					npos -= volume[dim+1];
				
				neighbors[2*D*i + c] = npos;
				c++;
			}
		}
	}


	// create 4 seed values for each thread:
	unsigned seeds[4*N/2];
	for(unsigned i=0; i<4*N/2; i++)
	{
		 seeds[i] = static_cast<unsigned>(4294967295 * drand48());
	}


	// calculate energy (Potts model)
	int E = 0;
	for(unsigned i=0; i<N; i++)	
	{
		for(unsigned j=0; j<2*D; j++)
		{
			if(spins[i] == spins[neighbors[2*D*i + j]])
				E--;
		}
	}
	E /= 2; // count each interaction only once


	// copy seeds to GPU:
	int *devPtrSeeds;
	hipMalloc((void**)&devPtrSeeds, sizeof(seeds));
	hipMemcpy(devPtrSeeds, &seeds, sizeof(seeds), hipMemcpyHostToDevice);

	// copy spins to GPU:
	unsigned short *devPtrSpins;
	hipMalloc((void**)&devPtrSpins, sizeof(spins));
	hipMemcpy(devPtrSpins, &spins, sizeof(spins), hipMemcpyHostToDevice);

	// copy neighborhood table to GPU:
	unsigned *devPtrNeighbors;
	hipMalloc((void**)&devPtrNeighbors, sizeof(neighbors));
	hipMemcpy(devPtrNeighbors, &neighbors, sizeof(neighbors), hipMemcpyHostToDevice);

	// copy white ids to GPU:
	unsigned *devPtrWhite;
	hipMalloc((void**)&devPtrWhite, sizeof(white));
	hipMemcpy(devPtrWhite, &white, sizeof(white), hipMemcpyHostToDevice);

	// copy black ids to GPU:
	unsigned *devPtrBlack;
	hipMalloc((void**)&devPtrBlack, sizeof(black));
	hipMemcpy(devPtrBlack, &black, sizeof(black), hipMemcpyHostToDevice);

	// each block calculates energy difference to initial state:
	int energyDifferences[N_BLOCKS];
	for(unsigned i=0; i<N_BLOCKS; i++)
		energyDifferences[i] = 0;

	int *devPtrEnergyDifferences;
	hipMalloc((void**)&devPtrEnergyDifferences, sizeof(energyDifferences));
	hipMemcpy(devPtrEnergyDifferences, &energyDifferences, sizeof(energyDifferences), hipMemcpyHostToDevice);

	// each block calculates block's magnetization:
	int magnetization[N_BLOCKS];
	int *devPtrMagnetization;
	hipMalloc((void**)&devPtrMagnetization, sizeof(magnetization));


	//std::cout << 'T_Potts' << '\t' << 'T_Ising' << '\t' << '<E_Potts/N>' << '\t' << '-log(-mE_Potts/N)' 
	//<< '\t' << 'C_Potts/N' << '\t' << '<E_Ising/N>' << '\t' << '-log(-mE_Ising/N)' << '\t' << 'C_Ising/N' <<'\t' << 'Chi' << '\t' << 'U4' << std::endl;

	int E_before_simulation = E;
	long long M = 0;	// magnetization

	for(float T=0.75; T<=3.76; T+=0.0125)
	{
	//	long long sum_E 	= 0;
	//	long long sum_EE 	= 0;
		double sum_e 		= 0;
		double sum_ee		= 0;
		double sum_m 		= 0;
		double sum_mm 		= 0;
		double sum_mmmm 	= 0;

		for(unsigned i=0; i<N_SWEEPS; i++)
		{
			#ifndef CONTRT
			// White spins:
			runMetropolis<<<N_BLOCKS, N_THREADS_PER_BLOCK>>>(devPtrSeeds, devPtrSpins, devPtrNeighbors, devPtrWhite, 
								devPtrEnergyDifferences, devPtrMagnetization, 1.0f/T);

			// Black spins:
			runMetropolis<<<N_BLOCKS, N_THREADS_PER_BLOCK>>>(devPtrSeeds, devPtrSpins, devPtrNeighbors, devPtrBlack, 
									devPtrEnergyDifferences, devPtrMagnetization, 1.0f/T);
			#else
				// White spins:
			runMetropolis<<<N_BLOCKS, N_THREADS_PER_BLOCK>>>(devPtrSeeds, devPtrSpins, devPtrNeighbors, devPtrWhite, 
								devPtrEnergyDifferences, devPtrMagnetization, devPtrContrt,1.0f/T);

			// Black spins:
			runMetropolis<<<N_BLOCKS, N_THREADS_PER_BLOCK>>>(devPtrSeeds, devPtrSpins, devPtrNeighbors, devPtrBlack, 
									devPtrEnergyDifferences, devPtrMagnetization, devPtrContrt ,1.0f/T);
			#endif
			// Sum up energy after a thermalization time for mean energy value:
			if(i >= 0.2*N_SWEEPS)
			{
				// get energy changes from the GPU:
				hipMemcpy(&energyDifferences, devPtrEnergyDifferences, sizeof(energyDifferences), hipMemcpyDeviceToHost);

				// get magnetization from the GPU:
				hipMemcpy(&magnetization, devPtrMagnetization, sizeof(magnetization), hipMemcpyDeviceToHost);

				E = E_before_simulation;
				M = 0;
				for(unsigned t=0; t<N_BLOCKS; t++)	// take energy changes into account
				{
					E += energyDifferences[t];
					M += magnetization[t];
				}

				double m = static_cast<double>(M) / static_cast<double>(N);
				double e = static_cast<double>(E) / static_cast<double>(N);

				sum_e 	 += e;
				sum_ee 	 += e*e;
				sum_m    += m;
				sum_mm 	 += m*m;
				sum_mmmm += m*m*m*m;
			}
		}

		double beta = 1.0f / T;

		double mE_Potts  = sum_e / (0.8*N_SWEEPS);
		double mEE_Potts = sum_ee / (0.8*N_SWEEPS);
		double C_Potts 	 = beta*beta*(mEE_Potts - mE_Potts*mE_Potts);

		double T_Ising = 2*T;
		double mE_Ising = 2.0 * mE_Potts + 2.0;
		double C_Ising = C_Potts;

	//	mE_Potts /= static_cast<double>(N);
	//	mE_Ising /= static_cast<double>(N);
		C_Potts *= static_cast<double>(N);
		C_Ising *= static_cast<double>(N);

		double mM    = static_cast<double>(sum_m) / (0.8*N_SWEEPS);
		double mMM 	 = static_cast<double>(sum_mm) / (0.8*N_SWEEPS);
		double mMMMM = static_cast<double>(sum_mmmm) / (0.8*N_SWEEPS);

		// Binder Parameter:
		double chi = beta*(mMM - mM*mM);
		double U4 = 1.0 - mMMMM / (3.0 * mMM*mMM);

		std::cout<<T<<"\t"<<T_Ising<<"\t"<<mE_Potts<<"\t"<<-log10(-mE_Potts)<<"\t"<<C_Potts<<"\t"<<mE_Ising<<"\t"<<-log10(-mE_Ising)<<"\t"<<C_Ising<<"\t"<<chi<< "\t" << mM <<"\t"<<U4<<"\n";
	}

	hipFree(devPtrSeeds);
	hipFree(devPtrSpins);
	hipFree(devPtrNeighbors);
	hipFree(devPtrWhite);
	hipFree(devPtrBlack);
	hipFree(devPtrEnergyDifferences);


    #ifdef CONTRT
        cudaFree(devPtrContrt);
    #endif

	return 0;
}
