
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <ctime>
#include <cmath>

#define L 8 	// lattice length; must be even number!
#define D 3 	// dimensions
#define N 512   // number of spins on square lattice (N=L^D)
#define J_IN 	25.00
#define J_OU 	100.00
#define J_PB   -300.00
#define N_SWEEPS	100000
#define H_max 200
#define H_min -200

/* Distribute N/2 spins over threads and blocks:
   since we do "black" and "white" spins seperately, we only need half the threads:
   N_BLOCKS * N_THREADS_PER_BLOCK = N/2 */
#define N_BLOCKS		8   // number of blocks
#define N_THREADS_PER_BLOCK	32  // number of threads per block


/*
Generator of random numbers Tausworthe in the device:
	Three-step generator with period 2^88.

Keyword Arguments:
z1: unsigned (first step storage, is a random numbert)
z2: unsigned (second step storage, is a random numbert)
z3: unsigned (third step storage, is a random numbert)
*/
__device__ unsigned Tausworthe88(unsigned &z1, unsigned &z2, unsigned &z3)
{
	unsigned b = (((z1 << 13) ^ z1) >> 19);
	z1 = (((z1 & 4294967294) << 12) ^ b);

	b = (((z2 << 2) ^ z2) >> 25);
	z2 = (((z2 & 4294967288) << 4)  ^ b);

	b = (((z3 << 3) ^ z3) >> 11);
	z3 = (((z3 & 4294967280) << 17) ^ b);

	return z1 ^ z2 ^ z3;
}


/*
Generator of random numbers LCRNG in the device:
	Linear congruential random number generators with period 2^32.

Keyword Arguments:
z: unsigned (is a random numbert)
*/
__device__ unsigned LCRNG(unsigned &z)  
{  
	const unsigned a = 1664525, c = 1013904223;
	return z = a * z + c;
}


/*
Combination of a Tausworthe generator with a LCRNG, esulting in a 
generator with a period of about 2^120.

Keyword Arguments:
z1: unsigned (is a random numbert)
z2: unsigned (is a random numbert)
z3: unsigned (is a random numbert)
z: unsigned (is a random numbert)
*/
__device__ float TauswortheLCRNG(unsigned &z1, unsigned &z2, unsigned &z3, unsigned &z)
{
	// combine both generators and normalize 0...2^32 to 0...1
	return (Tausworthe88(z1, z2, z3) ^ LCRNG(z)) * 2.3283064365e-10;
}



/* 
Metropolis algorithm in the device. For this a decomposition was used in chessboard.

Keyword Arguments:
seeds: int * (random number generator seeds)
spins: int * (spin list)
neighbors: unsigned * (list of neighbors of a given spin)
spinIdList: unsigned * (PONER QUE ES ESTO)
*/

__global__ void runMetropolis(int *seeds, int* spins, unsigned* neighbors, unsigned* spinIdList, 
						int* energyDifferences, int* magnetization, bool* contrt, float beta, int H)
{
	const unsigned id = blockDim.x*blockIdx.x + threadIdx.x;

	// spin id from list of black or white spins:
	const unsigned spinId = spinIdList[id];

	// get seed values:
	unsigned z1 = seeds[4*id    ];	// Tausworthe seeds
	unsigned z2 = seeds[4*id + 1];
	unsigned z3 = seeds[4*id + 2];
	unsigned z  = seeds[4*id + 3];	// LCRNG seed

	// energy differences for this block:
	__shared__ int deltaE[N_THREADS_PER_BLOCK];
	__shared__ int blockSpins[N_THREADS_PER_BLOCK];
	deltaE[threadIdx.x] = 0;

	int spinstate = spins[spinId];	// get spin state from DRAM
	int nb[2*D];						// neighbor states

	bool conex = contrt[spinId];
    bool nl[2 * D];
    
	
	// get neighbor states:
	for(unsigned n=0; n<2*D; n++){
		nb[n] = spins[neighbors[2 * D * spinId + n]];
		nl[n] = contrt[neighbors[2 * D * spinId + n]];
        
	}

	// propose random new spin state:
	int newstate = - spinstate;

	// energy difference: E'-E
	int E_before = 0;
	int E_after  = 0;

	for(int n=0; n<2*D; n++){
		
		if(n != 4 && n != 5)
			E_before += J_IN * spinstate * nb[n];
		else{

			if (n == 4 && conex)
				E_before += J_PB * spinstate * nb[n];
			else if (n == 5 && nl[5])
				E_before += J_PB * spinstate * nb[n];
			else
				E_before += J_OU * spinstate * nb[n];
		}

		if(n != 4 && n != 5)
			E_after += J_IN * newstate * nb[n];
		else{

			if (n == 4 && conex)
				E_after += J_PB * newstate * nb[n];
			else if (n == 5 && nl[5])
				E_before += J_PB * newstate * nb[n];
			else
				E_after += J_OU * newstate * nb[n];
		}
	}

	E_before += H * spinstate;
	E_after  += H * newstate;

	// acceptance probability:
	float dE = __int2float_rn(E_before - E_after);
	float pAccept = __expf(-beta*dE);

	if(TauswortheLCRNG(z1, z2, z3, z) <= pAccept)
	{
		spins[spinId] = newstate;   // flip spin
		spinstate = newstate;

		deltaE[threadIdx.x] = E_before - E_after;	// note energy difference
	}

	// remember locally in block for calculation of magnetization:
	blockSpins[threadIdx.x] = spinstate;

	// store new seed values in DRAM:
	seeds[4*id    ] = z1;	// Tausworthe seeds
	seeds[4*id + 1] = z2;
	seeds[4*id + 2] = z3;
	seeds[4*id + 3] = z;	// LCRNG seed


	__syncthreads();


	// sum up this block's energy delta and magnetization:
	if(threadIdx.x == 0)
	{
		int blockEnergyDiff = 0;
		int M = 0;
		

		for(unsigned i=0; i<blockDim.x; i++)
		{
			blockEnergyDiff += deltaE[i];

			if(blockSpins[i] == -1){
				M -= 1;
				
			}
			else{
				M += 1;
				
			}
		}

		energyDifferences[blockIdx.x] += blockEnergyDiff;
		magnetization[blockIdx.x] = M;
		
	}
}



/******************************
 *  HOST FUNCTION (CPU PART)  *
 ******************************/

int main(int argc, char const **argv){
	int args = 1;

	char log_file[30];
	char hist[30];
	

	// variable responsible for storing concentration
    float x = (argc > args)?(atof(argv[args])):(0.23);
    args++;

    
    int T_cut = (argc > args)?(atoi(argv[args])):(0);
    args++;


	sprintf(log_file, "x=%0.2fT_cut=%0.2f.log", x, T_cut / 100.0);
	sprintf(hist, "Histeresis_x=%0.2f_T=%0.2f", x, T_cut / 100.0);
    std::ofstream lg(log_file);
	std::ofstream dt(hist);

	lg << "Author: William Carreras Oropesa" << std::endl;
	lg << "Simulation data" << std::endl;
	lg << "L = " << L << std::endl;
	lg << "D = " << D << std::endl;
	lg << "N = " << N << std::endl;
	lg << "J_IN = " << J_IN / 100.0 << std::endl;
	lg << "J_OU = " << J_OU / 100.0 << std::endl;
	lg << "J_PB = " << J_PB / 100.0 << std::endl;
	lg << "N_SWEEPS = " << N_SWEEPS << std::endl;

    srand48(time(NULL));

	// each spin has value 0,..,Q-1
	int spins[N];
	bool contrt[N];
    
	// calculate lattice volume elements:
	unsigned volume[D];

	for(unsigned i=0; i<=D; i++){
		
		if(i == 0)
			volume[i] = 1;
		else
			volume[i] = volume[i-1] * L;
	}	


	/* Determine the "checkerboard color" (black or white) for each site and
	   initialise lattice with random spin states: */
	unsigned w=0, b=0;
	unsigned white[N/2], black[N/2];	// store ids of white/black sites

	for(unsigned i=0; i<N; i++){
		
		// Sum of all coordinates even or odd? -> gives checkerboard color
		int csum = 0;
		for(int k=D-1; k>=0; k--)
			csum += ceil((i+1.0)/volume[k]) - 1;

		if((csum%2) == 0){	// white
		
			white[w] = i;
			w++;
		}
		else{				// black

			black[b] = i;
			b++;
		}

		// random spin state:
		spins[i] = 2 * floor(2 * drand48()) - 1;
        contrt[i] = false;
	}


	int n_ele = int(N * x);
    int n = 0;
    
    while(n < n_ele){
        
        unsigned index = static_cast<unsigned>(floor(N * drand48()));
        
        if(!contrt[index]){
           	
           	contrt[index] = true;
           	n++;
		}            	
    }


    bool* devPtrContrt;
    hipMalloc((void**)&devPtrContrt, sizeof(contrt));
    hipMemcpy(devPtrContrt, &contrt, sizeof(contrt), hipMemcpyHostToDevice);
    

	// neighborhood table:
	unsigned neighbors[2*D*N];

	// calculate neighborhood table:
	for(unsigned i=0; i<N; i++){
	
		int c=0;

		for(int dim=0; dim<D; dim++){	// dimension loop
		
			for(short dir=-1; dir<=1; dir+=2){		// two directions in each dimension
			
				// neighbor's id in spin list:
				int npos = i + dir * volume[dim];

				// periodic boundary conditions:
				int test = (i % volume[dim + 1]) + dir * volume[dim];

				if(test < 0)
					npos += volume[dim + 1];
				else if(test >= volume[dim + 1])
					npos -= volume[dim + 1];
				
				neighbors[2 * D * i + c] = npos;
				c++;
			}
		}
	}


	// create 4 seed values for each thread:
	unsigned seeds[4 * N / 2];

	for(unsigned i=0; i<4*N/2; i++)
		seeds[i] = static_cast<unsigned>(4294967295 * drand48());

	// copy seeds to GPU:
	int *devPtrSeeds;
	hipMalloc((void**)&devPtrSeeds, sizeof(seeds));
	hipMemcpy(devPtrSeeds, &seeds, sizeof(seeds), hipMemcpyHostToDevice);

	// copy spins to GPU:
	int *devPtrSpins;
	hipMalloc((void**)&devPtrSpins, sizeof(spins));
	hipMemcpy(devPtrSpins, &spins, sizeof(spins), hipMemcpyHostToDevice);

	// copy neighborhood table to GPU:
	unsigned *devPtrNeighbors;
	hipMalloc((void**)&devPtrNeighbors, sizeof(neighbors));
	hipMemcpy(devPtrNeighbors, &neighbors, sizeof(neighbors), hipMemcpyHostToDevice);

	// copy white ids to GPU:
	unsigned *devPtrWhite;
	hipMalloc((void**)&devPtrWhite, sizeof(white));
	hipMemcpy(devPtrWhite, &white, sizeof(white), hipMemcpyHostToDevice);

	// copy black ids to GPU:
	unsigned *devPtrBlack;
	hipMalloc((void**)&devPtrBlack, sizeof(black));
	hipMemcpy(devPtrBlack, &black, sizeof(black), hipMemcpyHostToDevice);

	// each block calculates energy difference to initial state:
	int energyDifferences[N_BLOCKS];
	for(unsigned i=0; i<N_BLOCKS; i++)
		energyDifferences[i] = 0;

	int *devPtrEnergyDifferences;
	hipMalloc((void**)&devPtrEnergyDifferences, sizeof(energyDifferences));
	hipMemcpy(devPtrEnergyDifferences, &energyDifferences, sizeof(energyDifferences), hipMemcpyHostToDevice);

	// each block calculates block's magnetization:
	int magnetization_white[N_BLOCKS];
	int *devPtrMagnetization_white;
	hipMalloc((void**)&devPtrMagnetization_white, sizeof(magnetization_white));

	int magnetization_black[N_BLOCKS];
	int *devPtrMagnetization_black;
	hipMalloc((void**)&devPtrMagnetization_black, sizeof(magnetization_black));

	

	for(float T = 500.00; T >= T_cut; T -= 10.00){

		for(unsigned i=0; i<N_SWEEPS; i++){
			
			// White spins:
			runMetropolis<<<N_BLOCKS, N_THREADS_PER_BLOCK>>>(devPtrSeeds, devPtrSpins, devPtrNeighbors, devPtrWhite, 
								devPtrEnergyDifferences, devPtrMagnetization_white, devPtrContrt,1.0f/T, 0);

			// Black spins:
			runMetropolis<<<N_BLOCKS, N_THREADS_PER_BLOCK>>>(devPtrSeeds, devPtrSpins, devPtrNeighbors, devPtrBlack, 
								devPtrEnergyDifferences, devPtrMagnetization_black, devPtrContrt ,1.0f/T, 0);
		}
	}

	hipMemcpy(&spins, devPtrSpins, sizeof(spins), hipMemcpyDeviceToHost);


	// calculate energy (Potts model)
	int E = 0;

	for(unsigned i=0; i<N; i++){	
		
		for(unsigned j=0; j<2*D; j++){
		
			if(j != 4 && j !=5)
				E -= J_IN * spins[i] * spins[neighbors[2*D*i + j]];
			
			else{
				
				if(j == 4 && contrt[i])
					E -= J_PB * spins[i] * spins[neighbors[2*D*i + j]];
				else if (j == 5 && contrt[neighbors[2*D*i + j]])
					E -= J_PB * spins[i] * spins[neighbors[2*D*i + j]];
				else
					E -= J_OU * spins[i] * spins[neighbors[2*D*i + j]];
			}
		}
	}

	E /= 2; // count each interaction only once



	int E_before_simulation = E;
	long long M;	// magnetization


	for(int H = 0; H < H_max ; H++){
		
		double sum_e 		= 0;
		double sum_ee		= 0;
		double sum_m 		= 0;
		double sum_mm 		= 0;
		

		for(unsigned i=0; i<N_SWEEPS; i++){
			
			// White spins:
			runMetropolis<<<N_BLOCKS, N_THREADS_PER_BLOCK>>>(devPtrSeeds, devPtrSpins, devPtrNeighbors, devPtrWhite, 
								devPtrEnergyDifferences, devPtrMagnetization_white, devPtrContrt,1.0f/T_cut, H);

			// Black spins:
			runMetropolis<<<N_BLOCKS, N_THREADS_PER_BLOCK>>>(devPtrSeeds, devPtrSpins, devPtrNeighbors, devPtrBlack, 
								devPtrEnergyDifferences, devPtrMagnetization_black, devPtrContrt ,1.0f/T_cut, H);
			
			if(i >= 0.2*N_SWEEPS){
					
				// get energy changes from the GPU:
				hipMemcpy(&energyDifferences, devPtrEnergyDifferences, sizeof(energyDifferences), hipMemcpyDeviceToHost);

				// get magnetization from the GPU:
				hipMemcpy(&magnetization_white, devPtrMagnetization_white, sizeof(magnetization_white), hipMemcpyDeviceToHost);

				hipMemcpy(&magnetization_black, devPtrMagnetization_black, sizeof(magnetization_black), hipMemcpyDeviceToHost);
				
					

					E = E_before_simulation;
					M = 0;

					for(unsigned t=0; t<N_BLOCKS; t++){	// take energy changes into account
				
						E += energyDifferences[t];
						M += magnetization_white[t] + magnetization_black[t];
					}

					double m = static_cast<double>(M) / static_cast<double>(N);
					double e = static_cast<double>(E) / static_cast<double>(N);

					sum_e 	 += e;
					sum_ee 	 += e*e;
				
					sum_m 	 += m;
					sum_mm   += m*m;
			}
		}

		double beta = 1.0f / T_cut;

		double mE_Ising  = sum_e / (0.8*N_SWEEPS);
		double mEE_Ising = sum_ee / (0.8*N_SWEEPS);
		double C_Ising 	 = beta*beta*(mEE_Ising - mE_Ising*mE_Ising);

		C_Ising *= static_cast<double>(N);
		
		
		double m_M    = static_cast<double>(sum_m) / (0.8*N_SWEEPS);
		double m_MM   = static_cast<double>(sum_mm) / (0.8*N_SWEEPS);


		// magnetization:
		double chi_uniform = beta*(m_MM - m_M*m_M);
		

		dt << H / 100.0  << "\t" << m_M << "\t" << chi_uniform << std::endl;
	}

	for(int H = H_max; H > H_min ; H--){
		
		double sum_e 		= 0;
		double sum_ee		= 0;
		double sum_m 		= 0;
		double sum_mm 		= 0;
		

		for(unsigned i=0; i<N_SWEEPS; i++){
			
			// White spins:
			runMetropolis<<<N_BLOCKS, N_THREADS_PER_BLOCK>>>(devPtrSeeds, devPtrSpins, devPtrNeighbors, devPtrWhite, 
								devPtrEnergyDifferences, devPtrMagnetization_white, devPtrContrt,1.0f/T_cut, H);

			// Black spins:
			runMetropolis<<<N_BLOCKS, N_THREADS_PER_BLOCK>>>(devPtrSeeds, devPtrSpins, devPtrNeighbors, devPtrBlack, 
								devPtrEnergyDifferences, devPtrMagnetization_black, devPtrContrt ,1.0f/T_cut, H);
			
			if(i >= 0.2*N_SWEEPS){
					
				// get energy changes from the GPU:
				hipMemcpy(&energyDifferences, devPtrEnergyDifferences, sizeof(energyDifferences), hipMemcpyDeviceToHost);

				// get magnetization from the GPU:
				hipMemcpy(&magnetization_white, devPtrMagnetization_white, sizeof(magnetization_white), hipMemcpyDeviceToHost);

				hipMemcpy(&magnetization_black, devPtrMagnetization_black, sizeof(magnetization_black), hipMemcpyDeviceToHost);
				
					

					E = E_before_simulation;
					M = 0;

					for(unsigned t=0; t<N_BLOCKS; t++){	// take energy changes into account
				
						E += energyDifferences[t];
						M += magnetization_white[t] + magnetization_black[t];
					}

					double m = static_cast<double>(M) / static_cast<double>(N);
					double e = static_cast<double>(E) / static_cast<double>(N);

					sum_e 	 += e;
					sum_ee 	 += e*e;
				
					sum_m 	 += m;
					sum_mm   += m*m;
				}
			}

			double beta = 1.0f / T_cut;

			double mE_Ising  = sum_e / (0.8*N_SWEEPS);
			double mEE_Ising = sum_ee / (0.8*N_SWEEPS);
			double C_Ising 	 = beta*beta*(mEE_Ising - mE_Ising*mE_Ising);

			C_Ising *= static_cast<double>(N);
		
		
			double m_M    = static_cast<double>(sum_m) / (0.8*N_SWEEPS);
			double m_MM   = static_cast<double>(sum_mm) / (0.8*N_SWEEPS);


			// magnetization:
			double chi_uniform = beta*(m_MM - m_M*m_M);
		

			dt << H / 100.0  << "\t"  << m_M << "\t" << chi_uniform << std::endl;
	}

	for(int H = H_min; H <= H_max ; H++){
		
		double sum_e 		= 0;
		double sum_ee		= 0;
		double sum_m 		= 0;
		double sum_mm 		= 0;
		

		for(unsigned i=0; i<N_SWEEPS; i++){
			
			// White spins:
			runMetropolis<<<N_BLOCKS, N_THREADS_PER_BLOCK>>>(devPtrSeeds, devPtrSpins, devPtrNeighbors, devPtrWhite, 
								devPtrEnergyDifferences, devPtrMagnetization_white, devPtrContrt,1.0f/T_cut, H);

			// Black spins:
			runMetropolis<<<N_BLOCKS, N_THREADS_PER_BLOCK>>>(devPtrSeeds, devPtrSpins, devPtrNeighbors, devPtrBlack, 
								devPtrEnergyDifferences, devPtrMagnetization_black, devPtrContrt ,1.0f/T_cut, H);
			
			if(i >= 0.2*N_SWEEPS){
					
				// get energy changes from the GPU:
				hipMemcpy(&energyDifferences, devPtrEnergyDifferences, sizeof(energyDifferences), hipMemcpyDeviceToHost);

				// get magnetization from the GPU:
				hipMemcpy(&magnetization_white, devPtrMagnetization_white, sizeof(magnetization_white), hipMemcpyDeviceToHost);

				hipMemcpy(&magnetization_black, devPtrMagnetization_black, sizeof(magnetization_black), hipMemcpyDeviceToHost);
				
					

					E = E_before_simulation;
					M = 0;

					for(unsigned t=0; t<N_BLOCKS; t++){	// take energy changes into account
				
						E += energyDifferences[t];
						M += magnetization_white[t] + magnetization_black[t];
					}

					double m = static_cast<double>(M) / static_cast<double>(N);
					double e = static_cast<double>(E) / static_cast<double>(N);

					sum_e 	 += e;
					sum_ee 	 += e*e;
				
					sum_m 	 += m;
					sum_mm   += m*m;
				}
			}

			double beta = 1.0f / T_cut;

			double mE_Ising  = sum_e / (0.8*N_SWEEPS);
			double mEE_Ising = sum_ee / (0.8*N_SWEEPS);
			double C_Ising 	 = beta*beta*(mEE_Ising - mE_Ising*mE_Ising);

			C_Ising *= static_cast<double>(N);
		
		
			double m_M    = static_cast<double>(sum_m) / (0.8*N_SWEEPS);
			double m_MM   = static_cast<double>(sum_mm) / (0.8*N_SWEEPS);


			// magnetization:
			double chi_uniform = beta*(m_MM - m_M*m_M);
		

			dt << H / 100.0  << "\t" << m_M << "\t" << chi_uniform << std::endl;
	}

	

	hipFree(devPtrSeeds);
	hipFree(devPtrSpins);
	hipFree(devPtrNeighbors);
	hipFree(devPtrWhite);
	hipFree(devPtrBlack);
	hipFree(devPtrEnergyDifferences);
	hipFree(devPtrContrt);
	hipFree(devPtrMagnetization_white);
	hipFree(devPtrMagnetization_black);

	lg.close();
	dt.close();

    

	return 0;
}
